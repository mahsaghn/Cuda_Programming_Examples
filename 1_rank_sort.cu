
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 30;
const int blocksize = 16;

__global__ void vector_add(float *out, float *a, int n) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   int counter = 0;
   for(int i = 0; i < n; i ++){
        if(a[i]<a[tid] || (a[i] == a[tid] && tid>i)){
           counter++;
        }
    }
   out[counter] = a[tid];
}

int main(){
   float *a, *out;
   float *d_a, *d_out;

   a = (float*)malloc(sizeof(float) * N);
   out = (float*)malloc(sizeof(float) * N);
   for( int i =0; i<N; i++){
      a[i] = (N-i + 5)%(N-5);
   }

   hipMalloc((void**)&d_a, sizeof(float) * N);
   hipMalloc((void**)&d_out, sizeof(int) * N);
   hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

   vector_add<<<1,N>>>(d_out, d_a, N);
   hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
   
   for( int i =0; i<N; i++){
      printf("%f : %f\n",a[i], out[i]);
   }

   // Cleanup after kernel execution
   hipFree(d_a);
   hipFree(d_out);
   free(a);
   free(out);
}
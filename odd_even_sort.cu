
#include <hip/hip_runtime.h>
#include <stdio.h>
const int N = 24;
const int blocksize = 16;

__global__ void vector_sort(float *a, int n) {
   __shared__ float *sh_a;  
   sh_a = a;
   int id = blockIdx.x * blockDim.x + threadIdx.x;
   int tid = 0; 
   if(id < N-1)
   {
      for(int i = 0; i <= n; i++){
         
         if(i%2==0)
         {
            tid = id*2;
         }
         else
         {
            tid = id*2+1;
         }
         if(sh_a[tid] > sh_a[tid+1]){
            float h = sh_a[tid];
            sh_a[tid] = sh_a[tid+1];
            sh_a[tid+1] = h;
         }
         __syncthreads();
      }
   }
}

int main(){
   float *a;
   float *d_a;

   a = (float*)malloc(sizeof(float) * N);
   for( int i =0; i<N; i++){
      a[i] = (7*(N-i) + 5)%(N-5);
      printf("%d : %f\n",i,a[i]);
   }
   printf("_________\n");
   hipMalloc((void**)&d_a, sizeof(float) * N);
   hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

   vector_sort<<<1,N/2>>>(d_a, N);
   hipMemcpy(a, d_a, sizeof(float) * N, hipMemcpyDeviceToHost);
   
   for( int i =0; i<N; i++){
      printf("%d : %f\n",i,a[i]);
   }
   printf("\n");

   // Cleanup after kernel execution
   hipFree(d_a);
   free(a);
}
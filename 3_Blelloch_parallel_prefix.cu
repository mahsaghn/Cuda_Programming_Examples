
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
const int N = 32;
const int blocksize = 16;

__global__ void vector_sort(int *a, int n) {
   __shared__ int *sh_a;  
   __shared__ int round;
   sh_a = a;
   int powround=1;
   int id = blockIdx.x * blockDim.x + threadIdx.x;
   int tid=0;
   for(round = 1; round <= n; round++)
   {
      tid = id*2;
      tid +=1 ;
      powround *=2;
      if((tid+1)% powround==0)
      {
         int prev_id = tid - (int)(powround/2);
         if(prev_id>0){
            sh_a[tid] += sh_a[prev_id];
         }
      }
      __syncthreads();
   }
   powround = (int)powround/2;
   for(; powround >= 2;)
   {
      tid = powround* id + (powround/2)-1;
      if((N-1- (int)(powround/2)-tid)% powround == 0)
      {
         int prev_id = tid - (int)(powround/2);
         if(prev_id>0){
            sh_a[tid] += sh_a[prev_id];
         }
      }
      __syncthreads();
      powround /=2;
   }
}

int main(){
   int *a;
   int *d_a;

   a = (int*)malloc(sizeof(int) * N);
   for( int i =0; i<N; i++){
      a[i] = i;
      printf("%d : %d\n",i,a[i]);
   }
   printf("_________\n");
   hipMalloc((void**)&d_a, sizeof(int) * N);
   hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);

   vector_sort<<<1,N/2>>>(d_a, log2(N));
   hipMemcpy(a, d_a, sizeof(int) * N, hipMemcpyDeviceToHost);
   
   for( int i =0; i<N; i++){
      printf("%d : %d\n",i,a[i]);
   }
   printf("\n");

   // Cleanup after kernel execution
   hipFree(d_a);
   free(a);
}